#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <string>

#include "img_utils.hpp"

using namespace std;

__global__ void vecRGBtoG(float *imgIn, float *imgOut, int width, int height, int channels) {
  
  int x = blockIdx.x*blockDim.x + threadIdx.x; //col
  int y = blockIdx.y*blockDim.y + threadIdx.y; //row

  if (x < width && y < height) {
    int grayOffset = y * width + x;
    int rgbOffset = grayOffset*channels;
    float r = imgIn[rgbOffset];
    float g = imgIn[rgbOffset+1];
    float b = imgIn[rgbOffset+2];
    imgOut[grayOffset] = (0.21*r + 0.71*g + 0.07*b);
  }
      
}
 
int main(int argc, char **argv)
{
  if(argc!=3) {cout<<"Program takes two image filenames as parameters"<<endl;exit(3);}
  float *imgIn, *imgOut;
  int nCols, nRows, channels;

  imgIn = read_image_asfloat(argv[1],&nCols, &nRows, &channels);
  if(channels!=3){cout<<"Input image is not a colored image"<<endl;exit(4);}
  imgOut = (float *)calloc(nCols*nRows, sizeof(float));

  int size_in = sizeof(float)*nCols*nRows*channels;
  int size_out = size_in/channels;

  float *d_imgIn, *d_imgOut;
  hipMalloc((void **) &d_imgIn, size_in);
  hipMalloc((void **) &d_imgOut, size_out);

  hipMemcpy(d_imgIn, imgIn, size_in,hipMemcpyHostToDevice);

  dim3 DimGrid((nRows-1)/16 + 1, (nCols-1)/16+1, 1);
  dim3 DimBlock(16, 16, 1);
  //dim3 DimGrid((nRows-1)/32+ 1, (nCols-1)/32+1, 1);
  //dim3 DimBlock(32, 32, 1);
  
  vecRGBtoG<<<DimGrid,DimBlock>>>(d_imgIn, d_imgOut, nCols, nRows, channels);
  hipMemcpy(imgOut, d_imgOut, size_out, hipMemcpyDeviceToHost);

  write_image_fromfloat(argv[2], imgOut, nCols, nRows, 1);

  hipFree(d_imgIn); hipFree(d_imgOut); 


  return 0;
}
