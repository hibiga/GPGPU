
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

__global__ void kernel(int *a, int N) {
  int i=blockIdx.x*blockDim.x+threadIdx.x;

  /* 
  if (i > N) {
  printf("%d\n", i);
  }
  */
  
  if (i < N) {  //or il y a 33 blocks donc i depasse 4097 
  		//juste le premier thread du 33eme block est lu, le reste est en trop
  		//en testant i depasse N qui est égale a 4097 
  	
  	a[i]=i;
  }
  
}
//err = out of bounds 
  // donc i va trop loin par rapport a la taille du tab a[]
  // avec cuda-memcheck on voit que erreur sur le thread (63,0,0) au thread (32,0,0) 
  // dans le block (32,0,0)

int main() {
  
  int N=4097; 				//nombre d'element dans le tableau
  int threads=128;			//nombre de threads par blocks
  int blocks=(N+threads-1)/threads;	//calcul du nombre de blocks : arrondi au superieur
  int *a;

  hipMallocManaged(&a,N*sizeof(int));
  kernel<<<blocks,threads>>>(a, N);
  hipDeviceSynchronize();

  printf("\n   ICI   \n");
  for(int i=0;i<10;i++)
    printf("%d\n",a[i]);

  hipFree(a);

  cudaCheckError();
  return 0;
}
