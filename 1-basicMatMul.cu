#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "matmul_utils.hpp"


// Cuda kernel
__global__ void dgemm(float *A, float *B, float *C,
                      int numARows, int numAColumns, int numBRows, int numBColumns) {
  int x =blockIdx.x*blockDim.x + threadIdx.x; //col
  int y =blockIdx.y*blockDim.y + threadIdx.y; //row

  
  if (x < numBColumns && y < numARows) {
    int emp_c = x + y*numBColumns;
    C[emp_c]=0; //initialise
    for(int k = 0; k < numAColumns; k++) {
        int mat1 = k + y*numAColumns;
        int mat2 = x + k*numBColumns;
        C[emp_c] += A[mat1] * B[mat2];
    }
  }
}

int main(int argc, char **argv)
{
  if(argc!=4) {printf("Usage : %s [nb of rows for A] [nb of cols for A] [nb of cols for B]\n", argv[0]);exit(2);}
  //initilize a pseudo-random number generator
  srand(time(0));

  int numARows, numAColumns,numBRows, numBColumns,numCRows, numCColumns;
  // Read given dimensions
    //atoi : convert string to integer
  numARows = atoi(argv[1]);
  numAColumns = atoi(argv[2]);
  numBColumns  = atoi(argv[3]); 
  // Compute the remaining dimensions for given ones
  numBRows = numAColumns; 
  numCRows = numARows; 
  numCColumns = numBColumns; 
  printf("Matrix multiplication dimensions: [%d;%d] = [%d;%d] x [%d;%d]\n",
         numCRows, numCColumns, numARows, numAColumns, numBRows, numBColumns);
  // host pointers
  float *host_a, *host_b, *host_c;
  // Device pointers
  float *dev_a, *dev_b, *dev_c;
  
  int size = sizeof(float);
  int size_a = numAColumns*numARows*size;
  int size_b = numBColumns*numBRows*size;
  int size_c = numCColumns*numCRows*size;

  // Allocations on host
  host_a = (float *)calloc(numARows*numAColumns, size_a);
  host_b = (float *)calloc(numBRows*numBColumns, size_b);
  host_c = (float *)calloc(numCRows*numCColumns,size_c);

  // Initialize vectors
  init(host_a,host_b,numARows, numAColumns, numBRows, numBColumns);

  // Allocations on device
  hipMalloc((void **) &dev_a, size_a);
  hipMalloc((void **) &dev_b, size_b);
  hipMalloc((void **) &dev_c, size_c);

  // Copy from host to device
  hipMemcpy(dev_a, host_a, size_a,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, host_b, size_b,hipMemcpyHostToDevice);

  // Invoke kernel
  dim3 DimBlock(32,32,1);
  dim3 DimGrid((numARows-1)/DimBlock.x + 1, (numBColumns-1)/DimBlock.y+1, 1);

  // Initialize C device data
  hipMemset(dev_c, 0, numARows * numBColumns * size);

  // Call the kernel
  dgemm<<<DimGrid,DimBlock>>>(dev_a, dev_b, dev_c, numARows, numAColumns, numBRows, numBColumns);

  // Copy result from device to host
  hipMemcpy(host_c, dev_c, size_c,hipMemcpyDeviceToHost);

  // Check result
  check(host_a,host_b,host_c,numARows, numAColumns, numBRows, numBColumns);

  // Free device memory
  hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

  return 0;
}
