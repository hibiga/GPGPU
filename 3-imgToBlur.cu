#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <string>

#include "img_utils.hpp"

using namespace std;

__global__ void vecblurKernel(float* imgIn, float* imgOut, int width, int height, int channels) {
  int x =blockIdx.x*blockDim.x + threadIdx.x; 
  int y =blockIdx.y*blockDim.y + threadIdx.y; 
  if (y < height && x < width) {
    float sum_r = 0;
    float sum_g = 0;
    float sum_b = 0;
    float sum_grey = 0;
    int total = 0;
    int grey_o = y * width + x;
    int rgb_o = channels * grey_o;
    if (y < 3 || x < 3 || y > height - 4 || x > width - 4) {
      for (int blurCol = -3; blurCol < 4; blurCol++) {
        int inCol = x + blurCol;
        if(inCol > -1 && inCol < width) {
          total += 1;
          if(channels == 3){
            int idx = channels * (y * width + inCol) ;
            sum_r += imgIn[idx ];
            sum_g += imgIn[idx + 1];
            sum_b += imgIn[idx + 2];
          } else{
            int idx = y * width + inCol;
            sum_grey += imgIn[idx];
          }
        }
      }
      if(channels == 3){
        imgOut[rgb_o ] = sum_r/total;
        imgOut[rgb_o + 1] = sum_g/total;
        imgOut[rgb_o + 2] = sum_b/total;
      } else{
        imgOut[grey_o ] = sum_grey/total;
      }
    } else{
      for (int blurRow = -3; blurRow < 4; blurRow++) {
        for (int blurCol = -3; blurCol < 4; blurCol++) {
          int inRow = y + blurRow;
	  int inCol = x + blurCol;
	   if(inRow > -3 || inRow < height || inCol > -3 || inCol < width) {
	    total += 1;
	    int grey_v = inRow * width + inCol;
	    if(channels == 3){
	      int rgb_v = channels*grey_v;
	      sum_r += imgIn[rgb_v ]; 
	      sum_g += imgIn[rgb_v + 1];
	      sum_b += imgIn[rgb_v + 2]; 
	    } else{
	      sum_grey += imgIn[grey_v ];
	    }
          }
        } 
      }
      if(channels == 3){
        imgOut[rgb_o] = sum_r/total;
        imgOut[rgb_o + 1] = sum_g/total;
        imgOut[rgb_o + 2] = sum_b/total;
      } else{
        imgOut[grey_o ] = sum_grey/total;
      }
    }
  }
}

#define stream_nb 4

int main(int argc, char **argv)
{
 if(argc!=3) {cout<<"Program takes two image filenames as parameters"<<endl;exit(3);}
 float *imgIn, *imgOut;
 int nCols, nRows, channels;

 imgIn = read_image_asfloat(argv[1],&nCols, &nRows, &channels);
 if(channels!=3){cout<<"Input image is not a colored image"<<endl;exit(4);}
 imgOut = (float *)calloc(nCols*nRows*channels, sizeof(float));

// int stream_nb = nCols*nRows*channels;

 int size = nCols*nRows*channels; 
 int stream_size = size/stream_nb;
 
// int col = nCols/stream_nb;
 int row = nRows/stream_nb ;

 hipStream_t* stream;
 stream = (hipStream_t*)malloc(stream_size);
// hipStream_t stream[stream_nb];
 
 cout<<"Size stream "<<stream_size<<endl;

// memoire partage
// hipHostAlloc((void **) &imgIn, size*sizeof(float), hipHostMallocDefault);
 hipHostAlloc((void **) &imgOut, size*sizeof(float), hipHostMallocDefault);

 // float *d_imgIn, *d_imgOut;
 float *d_imgIn[stream_nb];
 float *d_imgOut[stream_nb];
 
// stream
 for (int i=0; i<stream_nb; i++){
    hipStreamCreate(&(stream[i]));
    hipMalloc((void **) &d_imgIn[i], (stream_size) * sizeof(float));
    hipMalloc((void **) &d_imgOut[i], (stream_size) * sizeof(float));
  }

  for (int i = 0; i < stream_nb; i++){
    int start = i * stream_size;
    hipMemcpyAsync(d_imgIn[i], imgIn + start, stream_size * sizeof(float), hipMemcpyHostToDevice, stream[i]);
  } 

 dim3 DimBlock(16, 16);
 dim3 DimGrid(ceil(nCols/DimBlock.x), ceil(nRows/DimBlock.y));
 
 for (int i = 0; i < stream_nb; i++){
    vecblurKernel<<<DimGrid, DimBlock, 0, stream[i]>>>(d_imgIn[i], d_imgOut[i], nCols, row, channels);
  } 
  
  auto err = hipGetLastError();
  cout<<"Done"<<endl;
  
 for (int i = 0; i < stream_nb; i++) {
    int start = i * stream_size;
    hipMemcpyAsync(imgOut + start, d_imgOut[i], stream_size * sizeof(float), hipMemcpyDeviceToHost, stream[i]);
  }
 
 hipDeviceSynchronize();
 write_image_fromfloat(argv[2], imgOut, nCols, nRows, channels);
 
 free(imgIn); hipHostFree(imgOut);

 for (int k=0; k<stream_nb; k++){
    hipFree(d_imgIn[k]); hipFree(d_imgOut[k]); 
 }
 free(stream);
 
 return 0;
}
