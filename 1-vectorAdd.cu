#include <stdio.h>
#include <hip/hip_runtime.h>

// Initialize host vectors
void init(float *a, float *b, int n) {
  for (int i=0; i < n; ++i) {
    a[i] = i;
    b[i] = n-i;
  }
}

// Check result correctness
void check(float *c, int n) {
  int i = 0;
  while (i < n && c[i] == n) {
    ++i;
  }
  if (i == n)
    printf("Ok\n");
  else
    printf("Non ok\n");
}


// Cuda kernel
__global__ void vecAddKernel(float *a, float *b, float *c, int n) {
  //@TODO@ : complete kernel code
  int i = threadIdx.x+blockDim.x*blockIdx.x;
  if (i<n) c[i] = a[i] + b[i];
}

void vecAdd(float *host_a, float *host_b, float *host_c, int n) {
  // Device pointers
  float *dev_a, *dev_b, *dev_c;
  int size = n * sizeof(float);

  // Allocations on device
  //@TODO@ : complete here
  hipMalloc((void **) &dev_a, size);
  hipMalloc((void **) &dev_b, size);
  hipMalloc((void **) &dev_c, size);

  // Copy from host to device
  //@TODO@ : complete here
  hipMemcpy(dev_a, host_a, size,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, host_b, size,hipMemcpyHostToDevice);

  // Invoke kernel
  //@TODO@ : complete here
  // droit a plus de thread que de block
  /*
  //force a utiliser un seul thread par block 
  dim3 DimGrid((n-1)/256+1,1,1);
  dim3 DimBlock(1,1,1);
  */
  /*
  //force a utiliser un seul block
  dim3 DimBlock(256,1,1);
  dim3 DimGrid(1,1,1); 
  */
  // plusieurs thread par blocks et plusieurs blocks 
  dim3 DimBlock(256,1,1);
  dim3 DimGrid(1+(n-1)/DimBlock.x,1,1); 

  vecAddKernel<<<DimGrid,DimBlock>>>(dev_a, dev_b, dev_c,n);

  // Copy result from device to host
  //@TODO@ : complete here
  hipMemcpy(host_c, dev_c, size,hipMemcpyDeviceToHost);

    // Free device memory
  hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
}

int main(int argc, char **argv)
{
  if(argc!=2) {printf("Give the vector size as first parameter\n");exit(2);}
  int n = atoi(argv[1]); 
  printf("Vector size is %d\n",n);
  // host pointers
  float *a, *b, *c;
  int size = n * sizeof(float);

  // Allocations on host
  //@TODO@ : complete here

  a = (float *)malloc(size);
  if (a==NULL) return 1;
  b = (float *)malloc(size);
  if (b==NULL) return 1;
  c = (float *)malloc(size);
  if (c==NULL) return 1;

  // Initialize vectors
  init(a,b,n);

  // Call function
  vecAdd(a,b,c,n);

  // Check result
  check(c,n);

  // Free host memory
  free(a); free(b); free(c);
  return 0;
}
