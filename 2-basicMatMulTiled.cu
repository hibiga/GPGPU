#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "matmul_utils.hpp"


// Cuda kernel
__global__ void MatMulTiled(float *A, float *B, float *C, int width) {

  const int TILE_SIZE = 32;

  __shared__ float As[TILE_SIZE][TILE_SIZE];
  __shared__ float Bs[TILE_SIZE][TILE_SIZE];
  
  int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
      
  int y = by * TILE_SIZE + ty; // row = ligne verticale
  int x = bx * TILE_SIZE + tx; // col = colonne horizontale
  float cij=0.0;

  for (int k = 0; k < (TILE_SIZE+width-1)/TILE_SIZE; k++) {
    if (k*TILE_SIZE + tx < width && y < width)
        As[ty][tx] = A[y * width + k * TILE_SIZE + tx];
    else 
    	As[ty][tx] = 0.0;
    if (k*TILE_SIZE + ty < width && x < width)
        Bs[ty][tx] = B[(k * TILE_SIZE + ty) * width + x];
    else
    	Bs[ty][tx] = 0.0;
    __syncthreads();

    for (int n = 0; n < TILE_SIZE; n++)
        cij += As[ty][n] * Bs[n][tx];

    __syncthreads();
    }
    
  if (y < width && x < width)
  	//C[((by*blockDim.y + ty)*width)+(bx*blockDim.x)+tx] = cij;
  	C[y * width + x] = cij;
}

int main(int argc, char **argv)
{
  if(argc!=4) {printf("Usage : %s [nb of rows for A] [nb of cols for A] [nb of cols for B]\n", argv[0]);exit(2);}
  //initilize a pseudo-random number generator
  srand(time(0));

  // Read given dimensions
  int width;
  width = atoi(argv[1]); 
  //tile_size = atoi(argv[1]); 
   
  printf("Matrix multiplication dimensions: [%d;%d] = [%d;%d] x [%d;%d]\n",
         width, width, width, width, width, width);
  // host pointers
  float *host_a, *host_b, *host_c;
  // Device pointers
  float *dev_a, *dev_b, *dev_c;

  int size = width*width*sizeof(float);
  int size_matrix = width*width;
  
  // Allocations on host
  host_a = (float *)calloc(size_matrix, sizeof(float));
  host_b = (float *)calloc(size_matrix, sizeof(float));
  host_c = (float *)calloc(size_matrix, sizeof(float));

  // Initialize vectors
  init(host_a,host_b,width, width, width, width);

  // Allocations on device
  hipMalloc((void **) &dev_a, size);
  hipMalloc((void **) &dev_b, size);
  hipMalloc((void **) &dev_c, size);

  // Copy from host to device
  hipMemcpy(dev_a, host_a, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, host_b, size, hipMemcpyHostToDevice);

  // Invoke kernel
  dim3 DimBlock(32,32);
  //dim3 DimGrid((width-1)/DimBlock.x + 1, (width-1)/DimBlock.x+1);
  dim3 DimGrid((width + DimBlock.x - 1) / DimBlock.x, (width+DimBlock.y-1)/DimBlock.y);

  // Initialize C device data
  hipMemset(dev_c, 0, size);

  // Call the kernel
  MatMulTiled<<<DimGrid,DimBlock>>>(dev_a, dev_b, dev_c, width);

  // Copy result from device to host
  hipMemcpy(host_c, dev_c, size, hipMemcpyDeviceToHost);

  // Check result
  check(host_a,host_b,host_c,width, width, width, width);

  // Free device memory
  hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

  return 0;
}

