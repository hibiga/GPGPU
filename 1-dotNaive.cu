#include <stdio.h>
#include <hip/hip_runtime.h>

// Initialize host vectors
float init(float *a, float *b, int n) {
  srand(0);
  float dot = 0.0;
  for (int i=0; i < n; ++i) {
    a[i] = rand()/((float) RAND_MAX);
    b[i] = rand()/((float) RAND_MAX);
    dot += a[i]*b[i];
  }
  return dot;
}

#define BLOCK_SIZE 1024
// Cuda kernel
__global__ void dotNaive(float *a, float *b, float *res, int n) {
  // @TODO@ : Complete here kernel code
	__shared__ float partialSum[2*BLOCK_SIZE];
	unsigned int t = threadIdx.x;
	unsigned int start = 2*blockIdx.x*blockDim.x;
	
	// creation memoire partagée
	if (start + t < n)
		partialSum[t] = a[start + t] * b[start + t];
	else 
		partialSum[t] = 0.0;
	
	if (start + blockDim.x+t < n)
		partialSum[blockDim.x+t] = a[start + blockDim.x+t] * b[start + blockDim.x+t];
  	else
  		partialSum[blockDim.x+t] = 0.0;
  		
  	// calcul
	for (unsigned int stride = blockDim.x; stride > 0; stride /= 2) {
		__syncthreads();
		if (t < stride)
			partialSum[t] += partialSum[t+stride];
	}
	res[blockIdx.x] = partialSum[0];

}
    	
int main(int argc, char **argv)
{
  if(argc!=2) {printf("Give the vector size as first parameter\n");exit(2);}
  int n = atoi(argv[1]);
  // condition qu'on peut enlever pour version global 
  // if(n<2*BLOCK_SIZE) {printf("Parameter value is too small\n");exit(2);}
  printf("Vector size is %d\n",n);
  
  // @TODO@ : Complete block number
  // int block_nb =  1; // (start with 1 in the first version)
  int block_nb = (n - 1)/(2*BLOCK_SIZE)+1;

  // host pointers
  float *host_a, *host_b, *host_res;
  // Device pointers
  float *dev_a, *dev_b, *dev_res;

  // Allocations on host
  host_a = (float *)calloc(n, sizeof(float));
  host_b = (float *)calloc(n, sizeof(float));
  host_res = (float *)calloc(block_nb, sizeof(float));

  // Initialize vectors
  float dot_true = init(host_a,host_b,n);

  // Allocations on device
  hipMalloc((void **) &dev_a, n*sizeof(float));
  hipMalloc((void **) &dev_b, n*sizeof(float));
  hipMalloc((void **) &dev_res, block_nb*sizeof(float));

  // Copy from host to device
  hipMemcpy(dev_a, host_a, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, host_b, n*sizeof(float), hipMemcpyHostToDevice);

  // Invoke kernel
  dotNaive<<<block_nb,BLOCK_SIZE>>>(dev_a, dev_b, dev_res, n);

  // Copy result from device to host
  hipMemcpy(host_res, dev_res, block_nb*sizeof(float), hipMemcpyDeviceToHost);

  // Final host reduction : cpu
  for(int i=1; i<block_nb; i++)
    host_res[0] += host_res[i];

  // Check result
  if(fabs(host_res[0]-dot_true)/dot_true<1e-4)
    printf("Result Ok : \n");
  else
    printf("Wrong result (%g)\n",fabs(host_res[0]-dot_true)/dot_true);

  // Free device memory
  hipFree(dev_a); hipFree(dev_b); hipFree(dev_res);
  // Free host memory
  free(host_a); free(host_b); free(host_res);
  return 0;
}
